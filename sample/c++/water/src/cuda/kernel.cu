#include "hip/hip_runtime.h"
#include <particle_simulator.hpp>
#include <water_params.h>

#include "user_defined_class.h"
#include "cuda_pointer.h"
#include "kernel.h"

enum{
  N_WALK_LIMIT = 1200,
  NI_LIMIT     = N_WALK_LIMIT * 1000,
  NJ_LIMIT     = N_WALK_LIMIT * 10000,
  WARP_SIZE = 32,
};

class EpiDev{
 public:
  float4 rm;
  int    w;
  int   id; // type
};
class EpjDev{
 public:
  float4 rm;
  int    id; // type
};
class ForceDev{
 public:
  float4 fcl;
  float4 flj;
  __device__
  void init(){
    fcl.x = fcl.y = fcl.z = fcl.w = 0.0;
    flj.x = flj.y = flj.z = flj.w = 0.0;
  }
};

inline __device__
ForceDev accumulate_lips_sw
(const float4 rmi, const int idi, const int ti,
 const float4 rmj, const int idj, const int tj,
 const float  rc,
 const float  rc2i,
 const float  rclj2,
 ForceDev f
){
  // LIPS constant
  const float alpha2 = 0.19578f*0.19578f;
  const float au[9] = {
       0.0125143224110408f,
      -0.603493863454666f,
      11.7355819865242f,
     -96.296895305654f,
     216.649868508398f,
    -197.409191110696f,
      59.9544311773618f,
      13.9564907382725f,
      -8.66620089071555f
  };
  const float af[9] = {
     2.f*au[0],
     4.f*au[1],
     6.f*au[2],
     8.f*au[3],
    10.f*au[4],
    12.f*au[5],
    14.f*au[6],
    16.f*au[7],
    18.f*au[8]
  };
  const float rc3i = rc2i*rc2i*rc;
  const float bound_ele_pot = rc*rc2i*1.296557;

  // LJ constant
  const float ce12 = 4.f*EPSILON_OXY*powf(SIGMA_OXY,12);
  const float ce06 = 4.f*EPSILON_OXY*powf(SIGMA_OXY, 6);
  const float cf12 = 12.f*ce12;
  const float cf06 =  6.f*ce06;

  const float dx = rmi.x - rmj.x;
  const float dy = rmi.y - rmj.y;
  const float dz = rmi.z - rmj.z;
  const float r2 = ((dx*dx) + dy*dy) + dz*dz;

  // remove molecule itself
  if((idi/3) == (idj/3) && r2 < rclj2) return f;

  const float r2c2 = r2 * rc2i;
  // remove outside cutoff radii
  if(r2c2 > 1.f) return f;
  const float rinv  = rsqrtf(r2);
  const float coef  = r2c2 - alpha2;
  const float coef2 = coef*coef;
  const float utmp = r2c2*(au[0] + r2c2*
			  (au[1] + r2c2*
			  (au[2] + r2c2*
			  (au[3] + r2c2*
			  (au[4] + r2c2*
			  (au[5] + r2c2*
			  (au[6] + r2c2*
			  (au[7] + r2c2*
			  (au[8])))))))));
  const float ftmp =      (af[0] + r2c2*
		          (af[1] + r2c2*
		          (af[2] + r2c2*
		          (af[3] + r2c2*
		          (af[4] + r2c2*
		          (af[5] + r2c2*
		          (af[6] + r2c2*
		          (af[7] + r2c2*
		          (af[8])))))))));
  float fcl = rmj.w * (rinv*rinv*rinv + 0.5f*rc3i*coef2*(6.f*utmp + coef*ftmp));
  float ucl = rmj.w * (rinv - 0.5f*rc2i*rc * coef * coef2 * utmp - bound_ele_pot);
  f.fcl.x += rmi.w*fcl*dx;
  f.fcl.y += rmi.w*fcl*dy;
  f.fcl.z += rmi.w*fcl*dz;
  f.fcl.w += rmi.w*ucl;
  // lj interaction
  if(r2 > rclj2 || ti*tj == 0) return f;
  const float r2i  = rinv*rinv;
  const float r6i  = r2i*r2i*r2i;
  const float r12i = r6i*r6i;
  const float flj  = (cf12 * r12i - cf06 * r6i)*r2i;
  const float ulj  =  ce12 * r12i - ce06 * r6i;
  f.flj.x += flj*dx;
  f.flj.y += flj*dy;
  f.flj.z += flj*dz;
  f.flj.w += ulj;
  return f;
}

inline __device__
float4 warp_reduce(const float4 val){
  float4 ret = val;
#if (__CUDA_ARCH__ >= 300)
  for(int mask=16;mask>0;mask/=2){
    ret.x += __shfl_xor(ret.x,mask);
    ret.y += __shfl_xor(ret.y,mask);
    ret.z += __shfl_xor(ret.z,mask);
    ret.w += __shfl_xor(ret.w,mask);
  }
#else
  __shared__ float4 sh[32];
  sh[threadIdx.x] = val;
  for(int mask=16;mask>0;mask/=2){
    if(threadIdx.x < mask){
      sh[threadIdx.x].x += sh[threadIdx.x+mask].x;
      sh[threadIdx.x].y += sh[threadIdx.x+mask].y;
      sh[threadIdx.x].z += sh[threadIdx.x+mask].z;
      sh[threadIdx.x].w += sh[threadIdx.x+mask].w;
    }
  }
  ret = sh[threadIdx.x];
#endif
  return ret;
}

__global__
void kernel_lips_sw
(const int      *j_disp,
 const EpiDev   *epi,
 const EpjDev   *epj,
       ForceDev *force,
 const float    rccl,
 const float    rclj)
{
  const int tid = threadIdx.x;
  const int i   = blockIdx.x;

  const float rc    = rccl;
  const float rci   = 1.f / rc;
  const float rc2i  = rci*rci;
  const float rclj2 = rclj*rclj;

  const float4 rmi = epi[i].rm;
  const int    iw  = epi[i].w;
  const int    idi = epi[i].id;
  const int    ti  = (idi%3==0) ? 1:0;
  //ForceDev f = force[i];
  ForceDev f;
  f.init();
  for(int j=j_disp[iw]+tid;j<j_disp[iw+1];j+=WARP_SIZE){
    const float4 rmj = epj[j].rm;
    const int idj = epj[j].id;
    const int tj = (idj%3==0)?1:0;
    f = accumulate_lips_sw
      (rmi,idi,ti,
       rmj,idj,tj,
       rc,rc2i,rclj2,f);
  }
  f.fcl = warp_reduce(f.fcl);
  f.flj = warp_reduce(f.flj);
  if(tid==0) force[i] = f;
}

static cudaPointer<EpiDev>   dev_epi;
static cudaPointer<EpjDev>   dev_epj;
static cudaPointer<ForceDev> dev_force;
static cudaPointer<int>      ij_disp;
static bool init_call = true;

PS::S32 DispatchKernel
(const PS::S32          tag,
 const PS::S32          n_walk,
 const EP              *epi[],
 const PS::S32          n_epi[],
 const EP              *epj[],
 const PS::S32          n_epj[]
 ){
    assert(n_walk <= N_WALK_LIMIT);
    if(init_call){
      int ndevice = 1;
      hipGetDeviceCount(&ndevice);
      hipSetDevice(PS::Comm::getRank()%ndevice);
      dev_epi  .allocate(NI_LIMIT);
      dev_epj  .allocate(NJ_LIMIT);
      dev_force.allocate(NI_LIMIT);
      ij_disp  .allocate(N_WALK_LIMIT+2);
      init_call = false;
    }
    ij_disp[0] = 0;
    for(int k=0; k<n_walk; k++){
      ij_disp[k+1] = ij_disp[k] + n_epj[k];
    }
    ij_disp[n_walk+1] = ij_disp[n_walk];
    assert(ij_disp[n_walk] < NJ_LIMIT);
    ij_disp.htod(n_walk + 2);
    int ni_tot = 0;
    int nj_tot = 0;
    for(int iw=0; iw<n_walk; iw++){
      PS::F64vec gc = 0.0;
      for(int i=0; i<n_epi[iw]; i++){
	dev_epi[ni_tot].rm.x = epi[iw][i].pos.x;
	dev_epi[ni_tot].rm.y = epi[iw][i].pos.y;
	dev_epi[ni_tot].rm.z = epi[iw][i].pos.z;
	dev_epj[ni_tot].rm.w = epj[iw][i].charge;
	dev_epi[ni_tot].w = iw;
	dev_epi[ni_tot].id= epi[iw][i].id;
	ni_tot++;
      }
      for(int j=0; j<n_epj[iw]; j++){
	dev_epj[nj_tot].rm.x  = epj[iw][j].pos.x;
	dev_epj[nj_tot].rm.y  = epj[iw][j].pos.y;
	dev_epj[nj_tot].rm.z  = epj[iw][j].pos.z;
	dev_epj[nj_tot].rm.w  = epj[iw][j].charge;
	dev_epj[nj_tot].id = epj[iw][j].id;
	nj_tot++;
      }
    }
    assert(ni_tot < NI_LIMIT);
    assert(nj_tot < NJ_LIMIT);
    int ni_tot_reg = ni_tot;
    if(ni_tot_reg % WARP_SIZE != 0){
      ni_tot_reg /= WARP_SIZE;
      ni_tot_reg++;
      ni_tot_reg *= WARP_SIZE;
    }
    assert(ni_tot_reg < NI_LIMIT);
    for(int i=ni_tot; i<ni_tot_reg; i++){
      dev_epi[i].w = n_walk;
    }
    dev_epi.htod(ni_tot_reg);
    dev_epj.htod(nj_tot);
    int nblocks  = ni_tot;
    int nthreads = WARP_SIZE;
    const float rccl = 28.f;
    const float rclj = 4.f * SIGMA_OXY;
    kernel_lips_sw <<<nblocks, nthreads>>>
      (ij_disp, dev_epi,dev_epj, dev_force,rccl,rclj);
    return 0;
}

PS::S32 RetrieveKernel(const PS::S32 tag,
                       const PS::S32 n_walk,
                       const PS::S32 ni[],
                       Force *force[])
{
    int ni_tot = 0;
    for(int k=0; k<n_walk; k++){
      ni_tot += ni[k];
    }
    dev_force.dtoh(ni_tot);

    for(int iw=0; iw<n_walk; iw++){
      for(int i=0; i<ni[iw]; i++){
	force[iw][i].acc = 0.0;
	force[iw][i].pot = 0.0;
      }
    }
    int n_cnt = 0;
    for(int iw=0; iw<n_walk; iw++){
      for(int i=0; i<ni[iw]; i++){
	force[iw][i].acc.x += dev_force[n_cnt].fcl.x;
	force[iw][i].acc.y += dev_force[n_cnt].fcl.y;
	force[iw][i].acc.z += dev_force[n_cnt].fcl.z;
	force[iw][i].pot   += 0.5*dev_force[n_cnt].fcl.w;

	force[iw][i].acc.x += dev_force[n_cnt].flj.x;
	force[iw][i].acc.y += dev_force[n_cnt].flj.y;
	force[iw][i].acc.z += dev_force[n_cnt].flj.z;
	force[iw][i].pot   += 0.5*dev_force[n_cnt].flj.w;
	n_cnt++;
      }
    }
    return 0;
}
