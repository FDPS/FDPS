#include "hip/hip_runtime.h"
//#include "class.hpp"
//#include "force.hpp"
#include<particle_simulator.hpp>
#include "cuda_pointer.h"
#include "force_gpu_cuda.hpp"

enum{
	N_THREAD_GPU = 32,
	N_WALK_LIMIT = 1000,
	NI_LIMIT     = N_WALK_LIMIT*1000,
	NJ_LIMIT     = N_WALK_LIMIT*10000,
};

struct EpiGPU{
	float3 pos;
	int    id_walk;
};

struct EpjGPU{
	float4 posm;
};

struct ForceGPU{
	float4 accp;
};

inline __device__ float4 dev_gravity(
		float  eps2,
		float3 ipos,
		float4 jposm,
		float4 accp)
{
	float dx = jposm.x - ipos.x;
	float dy = jposm.y - ipos.y;
	float dz = jposm.z - ipos.z;

	float r2   = eps2 + dx*dx + dy*dy + dz*dz;
	float rinv = rsqrtf(r2);
	float pij  = jposm.w * rinv;
	float mri3 = rinv*rinv * pij;

	accp.x += mri3 * dx;
	accp.y += mri3 * dy;
	accp.z += mri3 * dz;
	accp.w -= pij;

	return accp;
}

#if 0
__global__ void ForceKernel(
		const int2   * ij_disp,
		const EpiGPU * epi,
		const EpjGPU * epj, 
		ForceGPU     * force,
		const float    eps2)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	const float3 ipos = epi[tid].pos;
    const int j_head = ij_disp[epi[tid].id_walk  ].y;
    const int j_tail = ij_disp[epi[tid].id_walk+1].y;

	float4 accp = make_float4(0.f, 0.f, 0.f, 0.f);
    for(int j=j_head; j<j_tail; j++){
		float4 jposm = epj[j].posm;
		accp = dev_gravity(eps2, ipos, jposm, accp);
	}

	force[tid].accp = accp;
}
#else
__device__ float4 ForceKernel_1walk(
		float4       *jpsh,
		const float3  ipos,
		const int     id_walk,
		const int2   *ij_disp,
		const EpjGPU *epj, 
		float4        accp,
		const float   eps2)
{
    const int tid = threadIdx.x;
    const int j_head = ij_disp[id_walk  ].y;
    const int j_tail = ij_disp[id_walk+1].y;

	for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
		// __syncthreads();
		jpsh[tid] = ((float4 *)(epj + j)) [tid];
		// __syncthreads();

		if(j_tail-j < N_THREAD_GPU){
			for(int jj=0; jj<j_tail-j; jj++){
				accp = dev_gravity(eps2, ipos, jpsh[jj], accp);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				accp = dev_gravity(eps2, ipos, jpsh[jj], accp);
			}
		}
	}
	
	return accp;
}

__device__ float4 ForceKernel_2walk(
		float4        jpsh[2][N_THREAD_GPU],
		const float3  ipos,
		const int     id_walk,
		const int     iwalk0,
		const int     iwalk1,
		const int2   *ij_disp,
		const EpjGPU *epj, 
		float4        accp,
		const float   eps2)
{
	const int jbeg0 = ij_disp[iwalk0].y;
	const int jbeg1 = ij_disp[iwalk1].y;
	const int jend0 = ij_disp[iwalk0 + 1].y;
	const int jend1 = ij_disp[iwalk1 + 1].y;
	const int nj0   = jend0 - jbeg0;
	const int nj1   = jend1 - jbeg1;

	const int nj_longer  = nj0 > nj1 ? nj0 : nj1;
	const int nj_shorter = nj0 > nj1 ? nj1 : nj0;
	const int walk_longer= nj0 > nj1 ? 0 : 1;
	const int jbeg_longer = nj0 > nj1 ? jbeg0 : jbeg1;

	const int mywalk = id_walk==iwalk0 ? 0 : 1;

    const int tid = threadIdx.x;
	for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
		jpsh[0][tid] = ((float4 *)(epj + jbeg0 + j)) [tid];
		jpsh[1][tid] = ((float4 *)(epj + jbeg1 + j)) [tid];
		if(nj_shorter-j < N_THREAD_GPU){
			for(int jj=0; jj<nj_shorter-j; jj++){
				accp = dev_gravity(eps2, ipos, jpsh[mywalk][jj], accp);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				accp = dev_gravity(eps2, ipos, jpsh[mywalk][jj], accp);
			}
		}
	}
	for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
		jpsh[0][tid] = ((float4 *)(epj + jbeg_longer +  j)) [tid];
		int jrem = nj_longer - j;
		if(jrem < N_THREAD_GPU){
			for(int jj=0; jj<jrem; jj++){
				if(mywalk == walk_longer)
				accp = dev_gravity(eps2, ipos, jpsh[0][jj], accp);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				if(mywalk == walk_longer)
				accp = dev_gravity(eps2, ipos, jpsh[0][jj], accp);
			}
		}
	}

	return accp;
}

__device__ float4 ForceKernel_multiwalk(
		const float3  ipos,
		const int     id_walk,
		const int2   *ij_disp,
		const EpjGPU *epj, 
		float4        accp,
		const float   eps2)
{
    const int j_head = ij_disp[id_walk  ].y;
    const int j_tail = ij_disp[id_walk+1].y;

#if 1
    for(int j=j_head; j<j_tail; j++){
		float4 jposm = epj[j].posm;
		accp = dev_gravity(eps2, ipos, jposm, accp);
	}
#else
	int njmin = j_tail - j_head;
	njmin = min(njmin, __shfl_xor(njmin, 1));
	njmin = min(njmin, __shfl_xor(njmin, 2));
	njmin = min(njmin, __shfl_xor(njmin, 4));
	njmin = min(njmin, __shfl_xor(njmin, 8));
	njmin = min(njmin, __shfl_xor(njmin, 16));
	
	njmin &= 3;;
	for(int j=0; j<njmin; j+=4){
#pragma unroll 4
		for(int jj=0; jj<4; jj++){
			float4 jposm = epj[j_head + j + jj].posm;
			float4 jposm = jpf[jj];
			accp = dev_gravity(eps2, ipos, jposm, accp);
		}
	}
    for(int j=j_head+njmin; j<j_tail; j++){
		float4 jposm = epj[j].posm;
		accp = dev_gravity(eps2, ipos, jposm, accp);
	}
#endif
	return accp;
}

__global__ void ForceKernel(
		const int2   * ij_disp,
		const EpiGPU * epi,
		const EpjGPU * epj, 
		ForceGPU     * force,
		const float    eps2)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	float3 ipos    = epi[tid].pos;
	int    id_walk = epi[tid].id_walk;
	float4 accp    = make_float4(0.f, 0.f, 0.f, 0.f);


	int t_head = blockDim.x * blockIdx.x;
	int t_tail = t_head + N_THREAD_GPU - 1;
	int nwalk_in_block = 1 + (epi[t_tail].id_walk - epi[t_head].id_walk);

	__shared__ float4 jpsh[2][N_THREAD_GPU];

	if(1 == nwalk_in_block){
		accp = ForceKernel_1walk(jpsh[0], ipos, id_walk, ij_disp, epj, accp, eps2);
	} else if(2 == nwalk_in_block){
		// accp = ForceKernel_multiwalk(ipos, id_walk, ij_disp, epj, accp, eps2);
		int iwalk0 = epi[t_head].id_walk;
		int iwalk1 = epi[t_tail].id_walk;
		accp = ForceKernel_2walk(jpsh, ipos, id_walk, iwalk0, iwalk1, ij_disp, epj, accp, eps2);
	} else{
		accp = ForceKernel_multiwalk(ipos, id_walk, ij_disp, epj, accp, eps2);
	}
	force[tid].accp = accp;
}
#endif

static cudaPointer<EpiGPU>   dev_epi;
static cudaPointer<EpjGPU>   dev_epj;
static cudaPointer<ForceGPU> dev_force;
static cudaPointer<int2>     ij_disp;
static bool init_call = true;

PS::S32 DispatchKernelWithSP(
                             const PS::S32          tag,
                             const PS::S32          n_walk,
                             const FPGrav          *epi[],
                             const PS::S32          n_epi[],
                             const FPGrav          *epj[],
                             const PS::S32          n_epj[],
                             const PS::SPJMonopole *spj[],
                             const PS::S32          n_spj[]){
    assert(n_walk <= N_WALK_LIMIT);
    if(init_call){
		dev_epi  .allocate(NI_LIMIT);
		dev_epj  .allocate(NJ_LIMIT);
		dev_force.allocate(NI_LIMIT);
		ij_disp  .allocate(N_WALK_LIMIT+2);
		init_call = false;
    }
    const float eps2 = FPGrav::eps * FPGrav::eps;
    ij_disp[0].x = 0;
    ij_disp[0].y = 0;
    for(int k=0; k<n_walk; k++){
        ij_disp[k+1].x = ij_disp[k].x + n_epi[k];
        ij_disp[k+1].y = ij_disp[k].y + (n_epj[k] + n_spj[k]);
    }
    ij_disp[n_walk+1] = ij_disp[n_walk];

    assert(ij_disp[n_walk].x < NI_LIMIT);
    assert(ij_disp[n_walk].y < NJ_LIMIT);
    ij_disp.htod(n_walk + 2);

    int ni_tot_reg = ij_disp[n_walk].x;
    if(ni_tot_reg % N_THREAD_GPU){
        ni_tot_reg /= N_THREAD_GPU;
        ni_tot_reg++;
        ni_tot_reg *= N_THREAD_GPU;
    }

    int ni_tot = 0;
    int nj_tot = 0;
    for(int iw=0; iw<n_walk; iw++){
        for(int i=0; i<n_epi[iw]; i++){
            dev_epi[ni_tot].pos.x = epi[iw][i].pos.x;
            dev_epi[ni_tot].pos.y = epi[iw][i].pos.y;
            dev_epi[ni_tot].pos.z = epi[iw][i].pos.z;
            dev_epi[ni_tot].id_walk = iw;
            ni_tot++;
        }
        for(int j=0; j<n_epj[iw]; j++){
            dev_epj[nj_tot].posm.x  = epj[iw][j].pos.x;
            dev_epj[nj_tot].posm.y  = epj[iw][j].pos.y;
            dev_epj[nj_tot].posm.z  = epj[iw][j].pos.z;
            dev_epj[nj_tot].posm.w  = epj[iw][j].mass;
            nj_tot++;
        }
        for(int j=0; j<n_spj[iw]; j++){
            dev_epj[nj_tot].posm.x  = spj[iw][j].pos.x;
            dev_epj[nj_tot].posm.y  = spj[iw][j].pos.y;
            dev_epj[nj_tot].posm.z  = spj[iw][j].pos.z;
            dev_epj[nj_tot].posm.w  = spj[iw][j].getCharge();
            nj_tot++;
        }
    }
    for(int i=ni_tot; i<ni_tot_reg; i++){
        dev_epi[i].id_walk = n_walk;
    }

    dev_epi.htod(ni_tot_reg);
    dev_epj.htod(nj_tot);

    int nblocks  = ni_tot_reg / N_THREAD_GPU;
    int nthreads = N_THREAD_GPU;
    ForceKernel <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_epj, dev_force, eps2);

    return 0;
}

PS::S32 RetrieveKernel(const PS::S32 tag,
                       const PS::S32 n_walk,
                       const PS::S32 ni[],
                       FPGrav    *force[])
{
    int ni_tot = 0;
    for(int k=0; k<n_walk; k++){
        ni_tot += ni[k];
    }
    dev_force.dtoh(ni_tot);

    int n_cnt = 0;
    for(int iw=0; iw<n_walk; iw++){
        for(int i=0; i<ni[iw]; i++){
            force[iw][i].acc.x = dev_force[n_cnt].accp.x;
            force[iw][i].acc.y = dev_force[n_cnt].accp.y;
            force[iw][i].acc.z = dev_force[n_cnt].accp.z;
            force[iw][i].pot   = dev_force[n_cnt].accp.w;
            n_cnt++;
        }
    }
    return 0;
}
